// Your code with revisions

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define MAX_REACH 4

typedef struct {
    int x;
    int y;
} Position;

__global__ void findPath(char *grid, int grid_size, Position start, Position end, double *distances, Position *previous, bool *visited, Position *path, Position *queue) {
    int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (tid_x < grid_size && tid_y < grid_size) {
        for (int i = 0; i < grid_size; i++) {
            distances[tid_x * grid_size + i] = INFINITY;
            visited[tid_x * grid_size + i] = false;
        }

        distances[start.x * grid_size + start.y] = 0;

        int queue_size = 0;

        queue[queue_size] = start;
        queue_size++;

        while (queue_size > 0) {
            int min_index = 0;
            double min_cost = distances[queue[0].x * grid_size + queue[0].y];
            for (int i = 1; i < queue_size; i++) {
                double cost = distances[queue[i].x * grid_size + queue[i].y];
                if (cost < min_cost) {
                    min_index = i;
                    min_cost = cost;
                }
            }

            Position cur_position = queue[min_index];
            queue[min_index] = queue[--queue_size];

            if (cur_position.x == end.x && cur_position.y == end.y)
                break;

            if (visited[cur_position.x * grid_size + cur_position.y])
                continue;

            visited[cur_position.x * grid_size + cur_position.y] = true;

            for (int dx = -MAX_REACH; dx <= MAX_REACH; dx++) {
                for (int dy = -MAX_REACH; dy <= MAX_REACH; dy++) {
                    int x = cur_position.x + dx;
                    int y = cur_position.y + dy;

                    if (x < 0 || x >= grid_size || y < 0 || y >= grid_size)
                        continue;

                    if (grid[x * grid_size + y] == 'c' && !visited[x * grid_size + y]) {
                        double new_distance = distances[cur_position.x * grid_size + cur_position.y] + sqrtf(dx * dx + dy * dy);
                        if (new_distance < distances[x * grid_size + y]) {
                            distances[x * grid_size + y] = new_distance;
                            grid[x * grid_size + y] = 'C';
                            queue[queue_size].x = x;
                            queue[queue_size].y = y;
                            queue_size++;
                            previous[x * grid_size + y] = cur_position;
                        }
                    }
                }
            }
        }

        int path_size = 0;
        Position cur_position = end;
        while (!(cur_position.x == start.x && cur_position.y == start.y)) {
            path[path_size++] = cur_position;
            cur_position = previous[cur_position.x * grid_size + cur_position.y];
        }
        path[path_size++] = start;

        if (path_size < 3 || path[0].x != end.x || path[0].y != end.y || path[path_size - 1].x != start.x || path[path_size - 1].y != start.y) {
            return;
        } 
        
        // Display the shortest path
        // printf("Shortest path:\n");
        for (int i = path_size - 1; i > 0; i--) {
            grid[path[i].x * grid_size + path[i].y] = 'X';
        }
    } else {
        // printf("Invalid thread index: (%d, %d)\n", tid_x, tid_y);
    }
}


void displayGrid(char *grid, int grid_size) {
    printf("\n");
    for (int i = 0; i < grid_size; i++) {
        for (int j = 0; j < grid_size; j++) {
            if (grid[i * grid_size + j] == 'X')
                printf("\033[0;32mX \033[0m");
            else
                printf("%c ", grid[i * grid_size + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: %s <grid_size>\n", argv[0]);
        return 1;
    }

    int requested_size = atoi(argv[1]);
    if (requested_size <= 0) {
        printf("Invalid grid size.\n");
        return 1;
    }

    int grid_size = requested_size;

    srand(time(NULL));

    // Allocate memory for the grid on the host
    char *grid;
    gpuErrorCheck(hipMallocManaged(&grid, grid_size * grid_size * sizeof(char)));

    // Initialize the grid
    for (int i = 0; i < grid_size * grid_size; i++) {
        grid[i] = (rand() % 5 == 0 || i == 0 || i == grid_size * grid_size - 1) ? 'c' : '.';
    }

    // Allocate memory for other arrays on the host
    double *distances;
    gpuErrorCheck(hipMallocManaged(&distances, grid_size * grid_size * sizeof(double)));

    Position *previous;
    gpuErrorCheck(hipMallocManaged(&previous, grid_size * grid_size * sizeof(Position)));

    bool *visited;
    gpuErrorCheck(hipMallocManaged(&visited, grid_size * grid_size * sizeof(bool)));

    Position *path;
    gpuErrorCheck(hipMallocManaged(&path, grid_size * grid_size * sizeof(Position)));

    Position *queue;
    gpuErrorCheck(hipMallocManaged(&queue, grid_size * grid_size * sizeof(Position)));

    Position start = {0, 0};
    Position end = {grid_size - 1, grid_size - 1};

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0); // Assuming device 0

    printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);

    // Define block dimension based on grid size
    int block_dim = 32;  // Define an appropriate block dimension
    int cuda_grid_size = (grid_size + block_dim - 1) / block_dim;

    dim3 blockSize(block_dim, block_dim);
    dim3 gridSize(cuda_grid_size, cuda_grid_size);


    printf("Grid size: %d, Block size: %d\n", cuda_grid_size, block_dim);

    clock_t start_time = clock();
    findPath<<<gridSize, blockSize>>>(grid, grid_size, start, end, distances, previous, visited, path, queue);
    gpuErrorCheck(hipDeviceSynchronize());
    clock_t end_time = clock();

    if (grid_size < 50) {
        displayGrid(grid, grid_size);
    }

    // Calculate the path size
    int total_distance = 0;
    for (int i = grid_size * grid_size; i > 0; i--) {
        if (path[i].x == 0 && path[i].y == 0) {
            continue;
        }
        int dx = abs(path[i].x - path[i - 1].x);
        int dy = abs(path[i].y - path[i - 1].y);
        total_distance += dx+dy;
        if (path[i].x == end.x && path[i].y == end.y) {
            printf("x: %d, y: %d", path[i].x, path[i].y);
            break;
        } else {
            printf("x: %d, y: %d => ", path[i].x, path[i].y);
        }
    }

    printf("\n");

    if (total_distance == 0) {
        printf("No path found.\n");
    } else {
        printf("Path found. Length: %d\n", total_distance);
    }

    printf("Execution time: %f seconds\n", (double)(end_time - start_time) / CLOCKS_PER_SEC);

    // Free memory allocated on the host
    gpuErrorCheck(hipFree(grid));
    gpuErrorCheck(hipFree(distances));
    gpuErrorCheck(hipFree(previous));
    gpuErrorCheck(hipFree(visited));
    gpuErrorCheck(hipFree(path));
    gpuErrorCheck(hipFree(queue));

    return 0;
}
