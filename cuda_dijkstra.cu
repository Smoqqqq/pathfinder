#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#define MAX_REACH 4

typedef struct {
    int x;
    int y;
} Position;

__global__ void
findPath(char *grid, int grid_size, Position start, Position end, double *distances, Position *previous, bool *visited,
         Position *path, Position *queue) {
    int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (tid_x < grid_size && tid_y < grid_size) {
        // Initialize distances and visited arrays
        for (int i = 0; i < grid_size * grid_size; i++) {
            distances[i] = INFINITY;
            visited[i] = false;
        }

        // Initialisation de la distance et la file d'attente
        // TODO: passer sur le CPU
        distances[start.x * grid_size + start.y] = 0;
        int queue_size = 0;
        queue[queue_size++] = start;

        // Dijkstra
        while (queue_size > 0) {
            // On trouve le point le plus proche
            int min_index = 0;
            double min_distance = distances[queue[0].x * grid_size + queue[0].y];
            for (int i = 1; i < queue_size; i++) {
                double distance = distances[queue[i].x * grid_size + queue[i].y];
                if (distance < min_distance) {
                    min_index = i;
                    min_distance = distance;
                }
            }

            Position cur_position = queue[min_index];
            queue[min_index] = queue[--queue_size];

            if (cur_position.x == end.x && cur_position.y == end.y)
                break;

            if (visited[cur_position.x * grid_size + cur_position.y])
                continue;

            visited[cur_position.x * grid_size + cur_position.y] = true;

            for (int dx = -MAX_REACH; dx <= MAX_REACH; dx++) {
                for (int dy = -MAX_REACH; dy <= MAX_REACH; dy++) {
                    int x = cur_position.x + dx;
                    int y = cur_position.y + dy;

                    // Le voisin est il dans les limites du tableau
                    if (x >= 0 && x < grid_size && y >= 0 && y < grid_size) {
                        // Est ce que le voisin est une ville et n'a pas été visité
                        if (grid[x * grid_size + y] == 'c' && !visited[x * grid_size + y]) {
                            double new_distance =
                                    distances[cur_position.x * grid_size + cur_position.y] + sqrtf(dx * dx + dy * dy);
                            if (new_distance < distances[x * grid_size + y]) {
                                distances[x * grid_size + y] = new_distance;
                                grid[x * grid_size + y] = 'C';
                                queue[queue_size].x = x;
                                queue[queue_size].y = y;
                                queue_size++;
                                previous[x * grid_size + y] = cur_position;
                            }
                        }
                    }
                }
            }
        }

        int path_size = 0;
        Position cur_position = end;
        while (!(cur_position.x == start.x && cur_position.y == start.y)) {
            path[path_size++] = cur_position;
            cur_position = previous[cur_position.x * grid_size + cur_position.y];
        }
        path[path_size++] = start;

        // vériier que le chemin est correct
        if (path_size < 3 || path[0].x != end.x || path[0].y != end.y || path[path_size - 1].x != start.x ||
            path[path_size - 1].y != start.y) {
            return;
        }

        // Show the path on the grid
        for (int i = path_size - 1; i > 0; i--) {
            grid[path[i].x * grid_size + path[i].y] = 'X';
        }
    }
}

void displayGrid(char *grid, int grid_size) {
    printf("\n");
    for (int i = 0; i < grid_size; i++) {
        for (int j = 0; j < grid_size; j++) {
            if (grid[i * grid_size + j] == 'X')
                printf("\033[0;32mX \033[0m");
            else
                printf("%c ", grid[i * grid_size + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: %s <grid_size>\n", argv[0]);
        return 1;
    }

    int requested_size = atoi(argv[1]);
    if (requested_size <= 0) {
        printf("Invalid grid size.\n");
        return 1;
    }

    int grid_size = requested_size;

    srand(time(NULL));

    char *grid;
    gpuErrorCheck(hipMallocManaged(&grid, grid_size * grid_size * sizeof(char)));

    // Initialize the grid
    for (int i = 0; i < grid_size * grid_size; i++) {
        grid[i] = (rand() % 1 == 0 || i == 0 || i == grid_size * grid_size - 1) ? 'c' : '.';
    }

    double *distances;
    gpuErrorCheck(hipMallocManaged(&distances, grid_size * grid_size * sizeof(double)));

    Position *previous;
    gpuErrorCheck(hipMallocManaged(&previous, grid_size * grid_size * sizeof(Position)));

    bool *visited;
    gpuErrorCheck(hipMallocManaged(&visited, grid_size * grid_size * sizeof(bool)));

    Position *path;
    gpuErrorCheck(hipMallocManaged(&path, grid_size * grid_size * sizeof(Position)));

    Position *queue;
    gpuErrorCheck(hipMallocManaged(&queue, grid_size * grid_size * sizeof(Position)));

    Position start = {0, 0};
    Position end = {grid_size - 1, grid_size - 1};

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0); // Assuming device 0

    printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);

    int block_dim = 32;
    int cuda_grid_size = (grid_size + block_dim - 1) / block_dim;

    dim3 blockSize(block_dim, block_dim);
    dim3 gridSize(cuda_grid_size, cuda_grid_size);


    printf("Using block size: %d x %d\n", block_dim, block_dim);
    printf("Using grid size: %d x %d, threads per block: %d\n", cuda_grid_size, cuda_grid_size, block_dim * block_dim);
    printf("Total of threads: %d\n", cuda_grid_size * cuda_grid_size * block_dim * block_dim);
    printf("Nb of grid cells: %d\n", grid_size * grid_size);

    clock_t start_time = clock();
    findPath<<<gridSize, blockSize>>>(grid, grid_size, start, end, distances, previous, visited, path, queue);
    gpuErrorCheck(hipDeviceSynchronize());
    clock_t end_time = clock();

//    for (int i = 0; i < grid_size; i++) {
//        for (int j = 0; j < grid_size; j++) {
//            if (!visited[i * grid_size + j] && grid[i * grid_size + j] == 'c') {
//                printf("%d, %d was not visited\n", i, j);
//            }
//        }
//    }

    if (grid_size < 100) {
        displayGrid(grid, grid_size);
    }

    // Longueure du chemin
    int total_distance = 0;
    for (int i = grid_size * grid_size; i > 0; i--) {
        if (path[i].x == 0 && path[i].y == 0) {
            continue;
        }
        int dx = abs(path[i].x - path[i - 1].x);
        int dy = abs(path[i].y - path[i - 1].y);
        total_distance += dx + dy;
        if (path[i].x == end.x && path[i].y == end.y) {
            printf("x: %d, y: %d", path[i].x, path[i].y);
            break;
        } else {
            printf("x: %d, y: %d => ", path[i].x, path[i].y);
        }
    }

    printf("\n");

    if (total_distance == 0) {
        printf("No path found.\n");
    } else {
        printf("Path found. Length: %d\n", total_distance);
    }

    printf("Execution time: %f seconds\n", (double) (end_time - start_time) / CLOCKS_PER_SEC);

    gpuErrorCheck(hipFree(grid));
    gpuErrorCheck(hipFree(distances));
    gpuErrorCheck(hipFree(previous));
    gpuErrorCheck(hipFree(visited));
    gpuErrorCheck(hipFree(path));
    gpuErrorCheck(hipFree(queue));

    return 0;
}
